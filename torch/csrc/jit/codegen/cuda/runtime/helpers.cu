#include "hip/hip_runtime.h"

__device__ constexpr int ceilDiv(int a, int b) {
  return (a + b - 1) / b;
}

__device__ constexpr int alignBufferSize(int buffer, int size) {
  return (buffer + (size - 1)) & ~(size - 1);
}

__device__ double clamp(double x, double minv, double maxv) {
  return x < minv ? minv : (x > maxv ? maxv : x);
}

__device__ float clamp(float x, double minv, double maxv) {
  return x < minv ? minv : (x > maxv ? maxv : x);
}

__device__ double frac(double x) {
  return x - trunc(x);
}

__device__ float frac(float x) {
  return x - trunc(x);
}

__device__ double gelu(double x) {
  return x * normcdf(x);
}

__device__ float gelu(float x) {
  return x * normcdf(x);
}

__device__ double reciprocal(double x) {
  return 1 / x;
}

__device__ float reciprocal(float x) {
  return 1 / x;
}

__device__ double relu(double x) {
  return x <= 0 ? 0 : x;
}

__device__ float relu(float x) {
  return x <= 0 ? 0 : x;
}

__device__ double remainder(double a, double b) {
  auto mod = ::fmod(a, b);
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ float remainder(float a, float b) {
  auto mod = ::fmod(a, b);
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ double sigmoid(double x) {
  return 1 / (1 + exp(-x));
}

__device__ float sigmoid(float x) {
  return 1 / (1 + exp(-x));
}

__device__ double threshold(double x, double t, double v) {
  return x <= t ? v : x;
}

__device__ float threshold(float x, double t, double v) {
  return x <= t ? v : x;
}

__device__ double where(bool c, double a, double b) {
  return c ? a : b;
}

__device__ float where(bool c, float a, float b) {
  return c ? a : b;
}

__device__ double randLike(Philox rnd) {
  return uniform(rnd(), rnd());
}

__device__ float randLikef(Philox rnd) {
  return uniformf(rnd());
}

__device__ constexpr int64_t remainder(int64_t a, int64_t b) {
  auto mod = a % b;
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}

__device__ constexpr int remainder(int a, int b) {
  auto mod = a % b;
  if ((mod != 0) && ((b < 0) != (mod < 0)))
    mod += b;
  return mod;
}
